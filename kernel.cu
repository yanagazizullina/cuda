#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <ctime>

using namespace std;
#define BLOCK_SIZE 16
const unsigned int N = 2000;
float a[N][N];
float b[N][N];
float arrayGPU[N][N];
float arrayCPU[N][N];


__global__ void multipleGPU(float* a, float* b, int n, float* c)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	float sum = 0.0f;
	int ia = n * BLOCK_SIZE * by + n * ty;
	int ib = BLOCK_SIZE * bx + tx;
	int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	for (int k = 0; k < n; k++)
		sum += a[ia + k] * b[ib + k * n];
	c[ic + n * ty + tx] = sum;
}

int main()
{
	float timerGPU;
	//float timerCPU;
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			a[i][j] = rand() % 10 * sizeof(float);
			b[i][j] = rand() % 10 * sizeof(float);
		}
	}
	/*
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << setw(4) << a[i][j];
		}
		cout << endl;
	}

	printf("-----------------------------\n");

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << setw(4) << b[i][j];
		}
		cout << endl;
	}*/


	//Matrix multiplication by CPU
	//hipEventRecord(start, 0);
	clock_t start_s = clock();
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			arrayCPU[i][j] = 0;
			for (int k = 0; k < N; k++) {
				arrayCPU[i][j] += a[i][k] * b[k][j];
			}
		}
	}
	clock_t stop_s = clock();
	cout << "\n CPU time " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 << " msec" << endl;
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&timerCPU, start, stop);
	/*printf("--------------Multiple CPU---------------\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << setw(4) << arrayCPU[i][j];
		}
		cout << endl;
	}*/


	//Matrix multiplication by GPU
	int size = N * N * sizeof(float);
	float* da, * db, * dc;
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);
	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, size);
	hipMalloc((void**)&dc, size);
	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size, hipMemcpyHostToDevice);
	hipEventRecord(start, 0);
	multipleGPU << <blocks, threads >> > (da, db, N, dc);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerGPU, start, stop);

	hipDeviceSynchronize();
	hipMemcpy(arrayGPU, dc, size, hipMemcpyDeviceToHost);
	/*printf("--------------Multiple GPU---------------\n");

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << setw(4) << arrayGPU[i][j];
		}
		cout << endl;
	}*/
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	//printf("\n CPU time %f msec\n", timerCPU);
	printf("\n GPU time %f msec\n", timerGPU);
}
